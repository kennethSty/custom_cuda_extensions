#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <hipblas.h>
#include <ATen/cuda/HIPContext.h>

#define CEIL_DIV(a, b) ((a + b - 1) / b)

__global__ void bias_relu_k(float* input, const float* bias,int batch, int n) {
	/*
	 Performs elementwise bias addition and relu
	 */
	int x = blockIdx.x * blockDim.x + threadIdx.x; //feature index
        int y = blockIdx.y * blockDim.y + threadIdx.y; //row (batch) index

	bool is_valid_thread = x < n && y < batch;
        if (is_valid_thread) {
	    input[y * n + x] = fmaxf(0.0, input[y * n + x] + bias[x]);
	}		
}

__host__ void custom_dense_relu_layer(const float* input,
	        const float* weight,	
		const float* bias,
		float* output,
		dim3 gridDim, dim3 blockDim,
		int batch, int n,
		float alpha, float beta,
		hipblasHandle_t handle) {

        //Cublas expects col major, but input and weight are row major order
	//i.e. input and weight are transposed in col major representation
	//=> swap weight and input in Sgemm call. 
	hipblasSgemm(
		handle,
		HIPBLAS_OP_N, 
		HIPBLAS_OP_N,
		n, batch, n,
		&alpha,
		weight, n,
		input, n,
		&beta,
		output, n
	);

	bias_relu_k<<<gridDim, blockDim>>>(output, bias, batch, n);
}

__global__ void square_k(float* input, int batch, int n) {
	/*
	Squares each element of input in-place
	*/
        int x = blockIdx.x * blockDim.x + threadIdx.x; //feature index
	int y = blockIdx.y * blockDim.y + threadIdx.y; //row (batch) index
	bool is_valid_thread = y < batch && x < n; 

	if (is_valid_thread) {
	    //Pytorch stores tensor in row major order
	    input[y * n + x] = input[y * n + x] * input[y * n + x]; 
	}
}

__host__ void custom_dense_square_layer(const float* input,
	        const float* weight,	
		float* output,
		dim3 gridDim, dim3 blockDim,
		int batch, int n,
		float alpha, float beta,
		hipblasHandle_t handle) {

	hipblasSgemm(
		handle,
		HIPBLAS_OP_N, 
		HIPBLAS_OP_N,
		n, batch, n,
		&alpha,
		weight, n,
		input, n,
		&beta,
		output, n
	);

	square_k<<<gridDim, blockDim>>>(output, batch, n);
}

__global__ void outer_prod_k_opt(const float* input,
		float* output, int batch, int n) {
	/*
	Cuda kernel computing the outer product of the input vector in place.
	Assumes start of 2d grid with 'batch' number of blocks of size (n, n).
	Reason for using 2d: Task is inherently 2d -> one matrix per input vector.
	A thread computes the (i,j) element of the nxn matrix for
	the b-th vector out of 'batch' input vectors.
	*/
        int y = blockIdx.y * blockDim.y + threadIdx.y; //row index
	int x = blockIdx.x * blockDim.x + threadIdx.x; //col index
	int z = blockIdx.z; //batch index
	bool is_valid_thread = (x < n) && (y < n) && (z < batch);

	if (is_valid_thread) {
	    int batch_offset = z * n * n;
	    int row_offset = y * n;
	    float elem_y = input[z * n + y];
	    float elem_x = input[z * n + x];
	    output[batch_offset + row_offset + x] = elem_y * elem_x;
	}
}

torch::Tensor custom_forward_opt(torch::Tensor input, torch::Tensor weight1,
		torch::Tensor bias, torch::Tensor weight2) {
	
	//Setup kernel execution variables
	int batch = input.size(0);
	int n = input.size(1);
	
	const int threads_per_dim = 32;
	const int num_blocks_n = CEIL_DIV(n, threads_per_dim);
        const int num_blocks_batch = CEIL_DIV(batch, threads_per_dim);	

	dim3 blockDim(threads_per_dim, threads_per_dim);
	dim3 gridDimDense(num_blocks_n, num_blocks_batch);
        dim3 gridDimOut(num_blocks_n, num_blocks_n, batch);

        //Allocate output tensors on same device as input tensor	
       	auto options = input.options();
	auto out1 = torch::empty({batch, n}, options);
	auto out2 = torch::empty({batch, n}, options);
	auto out3 = torch::empty({batch, n, n}, options);
        
	//Init variables needed for cuda Sgemm and cuda kernels
	hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
	const float alpha = 1.0f, beta = 0.0f;

	//Layer 1: Dense Relu 
        custom_dense_relu_layer(
		input.data_ptr<float>(),
		weight1.data_ptr<float>(),
		bias.data_ptr<float>(),
		out1.data_ptr<float>(),
		gridDimDense, blockDim,
		batch, n, alpha, beta, handle
	);

	//Layer 2: Dense Square
        custom_dense_square_layer(
		out1.data_ptr<float>(),
		weight2.data_ptr<float>(),
		out2.data_ptr<float>(),
		gridDimDense, blockDim,
		batch, n, alpha, beta, handle
	);

	// Layer 3: Outer Product
        outer_prod_k_opt<<<gridDimOut, blockDim>>>(
		out2.data_ptr<float>(),
		out3.data_ptr<float>(),
		batch,
		n
    	);//automatically linked from kernels_naive.cu 

    	hipDeviceSynchronize();
    	return out3;
}
